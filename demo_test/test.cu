#include <hip/hip_runtime_api.h> // hipMalloc, hipMemcpy, etc.
#include <hip/hip_complex.h>        // hipDoubleComplex
#include <custatevec.h>       // custatevecApplyMatrix
#include <stdio.h>            // printf
#include <stdlib.h>           // EXIT_FAILURE

int main(void) {

   const int nIndexBits = 3;
   const int nSvSize    = (1 << nIndexBits);
   const int nTargets   = 1;
   const int nControls  = 2;
   const int adjoint    = 0;

   int targets[]  = {2};
   int controls[] = {0, 1};

   hipDoubleComplex h_sv[]        = {{ 0.0, 0.0}, { 0.0, 0.1}, { 0.1, 0.1},
                                    { 0.1, 0.2}, { 0.2, 0.2}, { 0.3, 0.3},
                                    { 0.3, 0.4}, { 0.4, 0.5}};
   hipDoubleComplex h_sv_result[] = {{ 0.0, 0.0}, { 0.0, 0.1}, { 0.1, 0.1},
                                    { 0.4, 0.5}, { 0.2, 0.2}, { 0.3, 0.3},
                                    { 0.3, 0.4}, { 0.1, 0.2}};
   hipDoubleComplex matrix[] = {{0.0, 0.0}, {1.0, 0.0},
                               {1.0, 0.0}, {0.0, 0.0}};


   hipDoubleComplex *d_sv;
   hipMalloc((void**)&d_sv, nSvSize * sizeof(hipDoubleComplex));

   hipMemcpy(d_sv, h_sv, nSvSize * sizeof(hipDoubleComplex),
              hipMemcpyHostToDevice);

   //--------------------------------------------------------------------------

   // custatevec handle initialization
   custatevecHandle_t handle;

   custatevecCreate(&handle);

   void* extraWorkspace = nullptr;
   size_t extraWorkspaceSizeInBytes = 0;

   // check the size of external workspace
   custatevecApplyMatrixGetWorkspaceSize(
       handle, HIP_C_64F, nIndexBits, matrix, HIP_C_64F,
       CUSTATEVEC_MATRIX_LAYOUT_ROW, adjoint, nTargets, nControls,
       CUSTATEVEC_COMPUTE_64F, &extraWorkspaceSizeInBytes);

   // allocate external workspace if necessary
   if (extraWorkspaceSizeInBytes > 0)
       hipMalloc(&extraWorkspace, extraWorkspaceSizeInBytes);

   // apply gate
   custatevecApplyMatrix(
       handle, d_sv, HIP_C_64F, nIndexBits, matrix, HIP_C_64F,
       CUSTATEVEC_MATRIX_LAYOUT_ROW, adjoint, targets, nTargets, controls,
       nullptr, nControls, CUSTATEVEC_COMPUTE_64F,
       extraWorkspace, extraWorkspaceSizeInBytes);

   // destroy handle
   custatevecDestroy(handle);

   //--------------------------------------------------------------------------

   hipMemcpy(h_sv, d_sv, nSvSize * sizeof(hipDoubleComplex),
              hipMemcpyDeviceToHost);

   bool correct = true;
   for (int i = 0; i < nSvSize; i++) {
       if ((h_sv[i].x != h_sv_result[i].x) ||
           (h_sv[i].y != h_sv_result[i].y)) {
           correct = false;
           break;
       }
   }

   if (correct)
       printf("example PASSED\n");
   else
       printf("example FAILED: wrong result\n");

   hipFree(d_sv);
   if (extraWorkspaceSizeInBytes)
       hipFree(extraWorkspace);

   return EXIT_SUCCESS;
}
